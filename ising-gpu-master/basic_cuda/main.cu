#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <chrono>
#include <fstream>
#include <getopt.h>
#include <iostream>
#include <string>

#include <hip/hip_fp16.h>
#include <hiprand.h>
#include <hipblas.h>

#include <hipcub/hipcub.hpp>
#define CUB_CHUNK_SIZE ((1ll<<31) - (1ll<<28))

#include "cudamacro.h"
#include "hip/hip_runtime.h"
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"
#include "thrust/transform_reduce.h"

#define THREADS  128

#define N_EQUILIBRIUM 20000
#define co std::cout <<
#define en << std::endl;
#define J0 1.0f




enum Color {BLACK, WHITE, GREEN};

struct saxpy_functor
{
    const float a;

    saxpy_functor(float _a) : a(_a) {}

    __host__ __device__
        float operator()(const float& x, const float& y) const {
            return (x-a) * (x - a);
        }
};

// square<T> computes the square of a number f(x) -> x*x
template <typename T>
struct calculation
{
  float a;
  __host__ __device__
    T operator()(const T& x) const {
      return (x-a) * (x-a);
  }
};

// Initialize lattice spins
__global__ void init_spins(signed char* lattice,
                           const float* __restrict__ randvals,
                           const long long nx,
                           const long long ny) {
  const long long  tid = static_cast<long long>(blockDim.x) * blockIdx.x + threadIdx.x;
  if (tid >= nx * ny) return;

  float randval = randvals[tid];
  signed char val = (randval < 0.5f) ? -1 : 1;
  lattice[tid] = val;
}

__global__ void copy_lattice(const signed char* __restrict__ lattice, signed char* extra_lattice, const long long nx,
                              const long long ny) {
  const long long tid = static_cast<long long>(blockDim.x) * blockIdx.x + threadIdx.x;
  const int i = tid / ny;
  const int j = tid % ny;
  
  if (i >= nx || j >= ny) return;

  extra_lattice[i*ny + j] = lattice[i*ny + j];
}


__host__ __device__ inline float sum(float x) {
  return x;
}

__global__ void initialize_spin_energy(float j_1, float j_2, float* spin_energy, Color color, 
                               const signed char* __restrict__ lattice,
                               const long long nx,
                               const long long ny) {
  const long long tid = static_cast<long long>(blockDim.x) * blockIdx.x + threadIdx.x;
  const int i = tid / ny;
  const int j = tid % ny;

  if (i >= nx || j >= ny) return;

  // Set stencil indices with periodicity
  int ipp = (i + 1 < nx) ? i + 1 : 0;
  int ip2 = (i + 2 < nx) ? i + 2 : i + 2 - nx;
  int inn = (i - 1 >= 0) ? i - 1: nx - 1;
  int in2 = (i - 2 >= 0) ? i - 2 : i - 2 + ny;
  int jpp = (j + 1 < ny) ? j + 1 : 0;
  int jnn = (j - 1 >= 0) ? j - 1: ny - 1;
  int j2 = (j == (ny-1)) ? jpp : j;
  int j3 = (j == 0) ? jnn : j;


  // Compute sum of nearest neighbor spins

  signed char nn_sum;
  nn_sum = j_1*(lattice[inn * ny + j] + lattice[ipp * ny + j]) +  // vizinho 1 vertical
                      j_2*(lattice[ip2 * ny + j] + lattice[in2 * ny + j]) +  // vizinho 2 vertical
                      J0*(lattice[i * ny + j2] + lattice[i * ny + j3]);   // vizinho 1 horizontal

  spin_energy[(i*ny + j)] = sum(nn_sum);
}

//template<bool is_black>
__global__ void update_lattice(float j_1, float j_2, float* spin_energy, Color color, signed char* lattice,
                               const float* __restrict__ randvals,
                               const float t,
                               const long long nx,
                               const long long ny) {
  const long long tid = static_cast<long long>(blockDim.x) * blockIdx.x + threadIdx.x;
  const int i = tid / ny;
  const int j = tid % ny;

  if ((j%3) != ((color + i) % 3)) {
    return;
  } else if (i >= nx || j >= ny) return;

  // Set stencil indices with periodicity
  int ipp = (i + 1 < nx) ? i + 1 : 0;
  int ip2 = (i + 2 < nx) ? i + 2 : i + 2 - nx;
  int inn = (i - 1 >= 0) ? i - 1: nx - 1;
  int in2 = (i - 2 >= 0) ? i - 2 : i - 2 + ny;
  int jpp = (j + 1 < ny) ? j + 1 : 0;
  int jnn = (j - 1 >= 0) ? j - 1: ny - 1;
  int j2 = (j == (ny-1)) ? jpp : j;
  int j3 = (j == 0) ? jnn : j;


  // Compute sum of nearest neighbor spins

  signed char nn_sum;
  nn_sum = j_1*(lattice[inn * ny + j] + lattice[ipp * ny + j]) +  // vizinho 1 vertical
                      j_2*(lattice[ip2 * ny + j] + lattice[in2 * ny + j]) +  // vizinho 2 vertical
                      J0*(lattice[i * ny + j2] + lattice[i * ny + j3]);   // vizinho 1 horizontal

  

  // Determine whether to flip spin
  signed char lij = lattice[i * ny + j];
  float acceptance_ratio = exp(-2.0f * lij * nn_sum / t);

  if (randvals[i*ny + j] < acceptance_ratio) { // se entrar significa que flipou
    lattice[i * ny + j] = -lij;
  }
}

// Write lattice configuration to file
void write_lattice(signed char *lattice_g, signed char *lattice_b, signed char *lattice_w, std::string filename, long long nx, long long ny) {
  printf("Writing lattice to %s...\n", filename.c_str());
  signed char *lattice_h, *lattice_g_h, *lattice_b_h, *lattice_w_h;
  lattice_h = (signed char*) malloc(nx * ny * sizeof(*lattice_h));
  lattice_g_h = (signed char*) malloc(nx * ny/3 * sizeof(*lattice_g_h));
  lattice_b_h = (signed char*) malloc(nx * ny/3 * sizeof(*lattice_b_h));
  lattice_w_h = (signed char*) malloc(nx * ny/3 * sizeof(*lattice_w_h));

  CHECK_CUDA(hipMemcpy(lattice_g_h, lattice_g, nx * ny/3 * sizeof(*lattice_g), hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(lattice_b_h, lattice_b, nx * ny/3 * sizeof(*lattice_b), hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(lattice_w_h, lattice_w, nx * ny/3 * sizeof(*lattice_w), hipMemcpyDeviceToHost));

  for (int i = 0; i < nx; i++) {
    for (int j = 0; j < ny/3; j++) {
        if ((i%3) == 0) {
            lattice_h[i*ny + 3*j] = lattice_b_h[i*ny/3 + j];
            lattice_h[i*ny + 3*j+1] = lattice_w_h[i*ny/3 + j];
            lattice_h[i*ny + 3*j+2] = lattice_g_h[i*ny/3 + j];
        } else if ((i%3) == 1) {
            lattice_h[i*ny + 3*j] = lattice_g_h[i*ny/3 + j];
            lattice_h[i*ny + 3*j+1] = lattice_b_h[i*ny/3 + j];
            lattice_h[i*ny + 3*j+2] = lattice_w_h[i*ny/3 + j];
        } else {
            lattice_h[i*ny + 3*j] = lattice_w_h[i*ny/3 + j];
            lattice_h[i*ny + 3*j+1] = lattice_g_h[i*ny/3 + j];
            lattice_h[i*ny + 3*j+2] = lattice_b_h[i*ny/3+j];
        }
    }
  }

  std::ofstream f;
  f.open(filename);
  if (f.is_open()) {
    for (int i = 0; i < nx; i++) {
      for (int j = 0; j < ny; j++) {
         f << (int)lattice_h[i * ny + j] << " ";
      }
      f << std::endl;
    }
  }
  f.close();

  free(lattice_h);
  free(lattice_b_h);
  free(lattice_w_h);
}

void write_values(char* filename, float t, float sh) {
  std::ofstream f;
  f.open(filename, std::ios::app);
  if (f.is_open()) {
    f << t << ", " << sh << " ";
    f << std::endl;
    
  }
  f.close();
}

void update(float j1, float j2, float* total_energy, signed char *lattice, float* randvals, hiprandGenerator_t rng, float t, long long nx, long long ny) {

  // Setup CUDA launch configuration
  int blocks = (nx * ny + THREADS - 1) / THREADS;

  // Update black
  //copy_lattice<<<blocks, THREADS>>>(lattice_b, extra_lattice, nx, ny/2);
  CHECK_CURAND(hiprandGenerateUniform(rng, randvals, nx*ny));
  update_lattice<<<blocks, THREADS>>>(j1, j2, total_energy, Color::BLACK, lattice, randvals, t, nx, ny);

  // Update white
  //copy_lattice<<<blocks, THREADS>>>(lattice_w, extra_lattice, nx, ny/2);
  CHECK_CURAND(hiprandGenerateUniform(rng, randvals, nx*ny));
  update_lattice<<<blocks, THREADS>>>(j1, j2, total_energy, Color::WHITE, lattice,  randvals, t, nx, ny);

  CHECK_CURAND(hiprandGenerateUniform(rng, randvals, nx*ny));
  update_lattice<<<blocks, THREADS>>>(j1, j2, total_energy, Color::GREEN, lattice, randvals, t, nx, ny);
}

static void usage(const char *pname) {

  const char *bname = rindex(pname, '/');
  if (!bname) {bname = pname;}
  else        {bname++;}

  fprintf(stdout,
          "Usage: %s [options]\n"
          "options:\n"
          "\t-x|--lattice-n <LATTICE_N>\n"
          "\t\tnumber of lattice rows\n"
          "\n"
          "\t-y|--lattice_m <LATTICE_M>\n"
          "\t\tnumber of lattice columns\n"
          "\n"
          "\t-w|--nwarmup <NWARMUP>\n"
          "\t\tnumber of warmup iterations\n"
          "\n"
          "\t-n|--niters <NITERS>\n"
          "\t\tnumber of trial iterations\n"
          "\n"
          "\t-a|--alpha <ALPHA>\n"
          "\t\tcoefficient of critical temperature\n"
          "\n"
          "\t-s|--seed <SEED>\n"
          "\t\tseed for random number generation\n"
          "\n"
          "\t-o|--write-lattice\n"
          "\t\twrite final lattice configuration to file\n\n",
          bname);
  exit(EXIT_SUCCESS);
}

void saxpy_fast(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
{
    // Y <- A * X + Y
    thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), saxpy_functor(A));
}

int main(int argc, char **argv) {

  // float alpha = atof(argv[1]);
  // float t = atof(argv[2]);
  // float t_end = atof(argv[3]);
  // float step = atof(argv[4]);
  // char* fileName = argv[5];
  // long long ny = atoll(argv[6]);
  // int niters = atoi(argv[7]);
  float alpha = 0.376f;
  float t = 0.6f;
  float j1 = (1-alpha)*J0;
  float j2 = -alpha*J0;
  char* fileName = "0.376_fim.txt";
  long long ny = 10;
  int niters = 100000;
  // Defaults
  long long nx = 240;
  //long long ny = 12;
  //float alpha = 0.1f;
  int nwarmup = N_EQUILIBRIUM;
  bool write = false;
  unsigned long long seed = 1234ULL;



  hiprandGenerator_t rng;
  CHECK_CURAND(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10));
  CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed(rng, seed));

  // Setup cuRAND generator
  
  float *randvals;
  CHECK_CUDA(hipMalloc(&randvals, (nx * ny) * sizeof(*randvals)));


  signed char *lattice;
  CHECK_CUDA(hipMalloc(&lattice, (nx * ny) * sizeof(*lattice)));

  


  int blocks = (nx * ny + THREADS - 1) / THREADS;
  CHECK_CURAND(hiprandGenerateUniform(rng, randvals, (nx*ny)));
  init_spins<<<blocks, THREADS>>>(lattice, randvals, nx, ny);



  thrust::device_vector<float> spin_energy(nx*ny);
  float *spin_energy_ptr = thrust::raw_pointer_cast(&spin_energy[0]);
  // initialize_spin_energy<<<blocks, THREADS>>>(spin_energy_ptr, Color::WHITE, lattice, nx, ny);
  // initialize_spin_energy<<<blocks, THREADS>>>(spin_energy_ptr, Color::BLACK, lattice, nx, ny);
  // initialize_spin_energy<<<blocks, THREADS>>>(spin_energy_ptr, Color::GREEN, lattice, nx, ny);

  thrust::device_vector<float> total_energy(niters);
  

  // Warmup iterations
  printf("Starting warmup...\n");
  for (int i = 0; i < nwarmup; i++) {
    update(j1, j2, spin_energy_ptr, lattice, randvals, rng, t, nx, ny);
  }
  

  CHECK_CUDA(hipDeviceSynchronize());

  printf("Starting trial iterations...\n");
  auto t0 = std::chrono::high_resolution_clock::now();
  for (int i = 0; i < niters; i++) {
    
    update(j1, j2, spin_energy_ptr, lattice, randvals, rng, t, nx, ny);
    
    
    initialize_spin_energy<<<blocks, THREADS>>>(j1, j2, spin_energy_ptr, Color::WHITE, lattice, nx, ny);
    // double tt = 
    total_energy[i] = thrust::reduce(spin_energy.begin(), spin_energy.end()) / (-2);
    // for (int i = 0; i < nx; i++) {
    //   for (int j = 0; j < ny; j++) {
    //     if (spin_energy[i*ny+j] != 0) {
    //       co spin_energy[i*ny+j] << " " << i << " " << j en;
    //     }
    //   }
    // }
    
    //std::cout << total_energy[i] << std::endl;
    if (i % 10000 == 0) printf("Completed %d/%d iterations...\n", i+1, niters);
  }
  float sum2 = thrust::reduce(total_energy.begin(), total_energy.end());
  float sum3 = sum2 / niters;
  co "sum2: " << sum2 en;
  sum2 /= niters;
  calculation<float> unary_op;
  unary_op.a = sum2;
  thrust::plus<float> binary_op;
  float init = 0;

  float variance = thrust::transform_reduce(total_energy.begin(), total_energy.end(), unary_op, 0,  binary_op);
  co "variance: " << variance en;

  variance /= niters;
  float specific_heat = variance / (t * t * nx * ny);
  write_values(fileName, t, specific_heat);

  CHECK_CUDA(hipDeviceSynchronize());
  auto t1 = std::chrono::high_resolution_clock::now();

  double duration = (double) std::chrono::duration_cast<std::chrono::microseconds>(t1-t0).count();
  printf("REPORT:\n");
  printf("\tnGPUs: %d\n", 1);
  printf("\ttemperature: %f * %f\n", alpha, t);
  printf("\tseed: %llu\n", seed);
  printf("\twarmup iterations: %d\n", nwarmup);
  printf("\ttrial iterations: %d\n", niters);
  printf("\tlattice dimensions: %lld x %lld\n", nx, ny);
  printf("\telapsed time: %f sec\n", duration * 1e-6);
  printf("\tupdates per ns: %f\n", (double) (nx * ny) * niters / duration * 1e-3);

  

  return 0;
}
