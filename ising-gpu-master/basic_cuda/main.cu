#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <chrono>
#include <fstream>
#include <getopt.h>
#include <iostream>
#include <string>
#include <time.h>

#include <hip/hip_fp16.h>
#include <hiprand.h>
#include <hipblas.h>

#include <hipcub/hipcub.hpp>
#define CUB_CHUNK_SIZE ((1ll<<31) - (1ll<<28))

#include "cudamacro.h"

#define TCRIT 2.26918531421f
#define THREADS  128

enum Color {BLACK, WHITE, GREEN};

// // Initialize lattice spins
// __global__ void init_spins(signed char* lattice,
//                            const float* __restrict__ randvals,
//                            const long long nx,
//                            const long long ny) {
//   const long long  tid = static_cast<long long>(blockDim.x) * blockIdx.x + threadIdx.x;
//   if (tid >= nx * ny) return;

//   float randval = randvals[tid];
//   signed char val = (randval < 0.5f) ? -1 : 1;
//   lattice[tid] = val;
// }

// Initialize lattice spins
__global__ void init_spins(signed char* lattice,
                           const float* __restrict__ randvals,
                           const long long nx,
                           const long long ny) {
  const long long  tid = static_cast<long long>(blockDim.x) * blockIdx.x + threadIdx.x;
  if (tid >= nx * ny) return;

  // float randval = randvals[tid];
  // signed char val = (randval < 0.5f) ? -1 : 1;
  lattice[tid] = 1;
}

__global__ void calculate_spin_energy(signed char* lattice,
                                 float* spin_energy,
                                  const long long nx,
                                  const long long ny,
                                  float j0, float j1, float j2) {
  const long long tid = static_cast<long long>(blockDim.x) * blockIdx.x + threadIdx.x;
  const int i = tid / ny;
  const int j = tid % ny;

  if (i >= nx || j >= ny) return;


  int ipp = (i + 1 < nx) ? i + 1 : 0;
  int inn = (i - 1 >= 0) ? i - 1: nx - 1;
  int jpp = (j + 1 < ny) ? j + 1 : 0;
  int jnn = (j - 1 >= 0) ? j - 1: ny - 1;
  int jp2 = (j + 2 < ny) ? j + 2 : j + 2 - ny;
  int jm2 = (j - 2 >= 0) ? j - 2 : j - 2 + ny;

  spin_energy[i*ny+j] = (-lattice[i*ny+j]) * (j0 * (lattice[inn*ny+j] + lattice[ipp*ny+j]) + j1*(lattice[i*ny+jpp] + lattice[i*ny+jnn])
      + j2 * (lattice[i*ny+jp2] + lattice[i*ny+jm2]));
}

template<bool is_black>
__global__ void update_lattice(enum Color color, signed char* lattice,
                               const float* __restrict__ randvals,
                               const float inv_temp,
                               const long long nx,
                               const long long ny, float j0, float j1, float j2) {
  const long long tid = static_cast<long long>(blockDim.x) * blockIdx.x + threadIdx.x;
  const int i = tid / ny;
  const int j = tid % ny;

  if (i >= nx || j >= ny) return;
  if ((j%3) != ((i+color)%3)) return;

  // Set stencil indices with periodicity
  int ipp = (i + 1 < nx) ? i + 1 : 0;
  int inn = (i - 1 >= 0) ? i - 1: nx - 1;
  int jpp = (j + 1 < ny) ? j + 1 : 0;
  int jnn = (j - 1 >= 0) ? j - 1: ny - 1;
  int jp2 = (j + 2 < ny) ? j + 2 : j + 2 - ny;
  int jm2 = ((j - 2) >= 0) ? j - 2 : j-2 + ny;
  

  // Compute sum of nearest neighbor spins
  float nn_sum = j0 * (lattice[inn*ny+j] + lattice[ipp*ny+j]) + j1*(lattice[i*ny+jpp] + lattice[i*ny+jnn])
      + j2 * (lattice[i*ny+jp2] + lattice[i*ny+jm2]);
  
  

  // Determine whether to flip spin
  signed char lij = lattice[i * ny + j];
  float acceptance_ratio = exp(-2.0f * inv_temp * nn_sum * lij);
  if (randvals[i*ny + j] < acceptance_ratio) {
    lattice[i * ny + j] = -lij;
  }
}

// Write lattice configuration to file
void write_lattice(signed char *lattice_b, signed char *lattice_w, std::string filename, long long nx, long long ny) {
  printf("Writing lattice to %s...\n", filename.c_str());
  signed char *lattice_h, *lattice_b_h, *lattice_w_h;
  lattice_h = (signed char*) malloc(nx * ny * sizeof(*lattice_h));
  lattice_b_h = (signed char*) malloc(nx * ny/2 * sizeof(*lattice_b_h));
  lattice_w_h = (signed char*) malloc(nx * ny/2 * sizeof(*lattice_w_h));

  CHECK_CUDA(hipMemcpy(lattice_b_h, lattice_b, nx * ny/2 * sizeof(*lattice_b), hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(lattice_w_h, lattice_b, nx * ny/2 * sizeof(*lattice_w), hipMemcpyDeviceToHost));

  for (int i = 0; i < nx; i++) {
    for (int j = 0; j < ny/2; j++) {
      if (i % 2) {
        lattice_h[i*ny + 2*j+1] = lattice_b_h[i*ny/2 + j];
        lattice_h[i*ny + 2*j] = lattice_w_h[i*ny/2 + j];
      } else {
        lattice_h[i*ny + 2*j] = lattice_b_h[i*ny/2 + j];
        lattice_h[i*ny + 2*j+1] = lattice_w_h[i*ny/2 + j];
      }
    }
  }

  std::ofstream f;
  f.open(filename);
  if (f.is_open()) {
    for (int i = 0; i < nx; i++) {
      for (int j = 0; j < ny; j++) {
         f << (int)lattice_h[i * ny + j] << " ";
      }
      f << std::endl;
    }
  }
  f.close();

  free(lattice_h);
  free(lattice_b_h);
  free(lattice_w_h);
}

void update(enum Color color, signed char *lattice, float* randvals, hiprandGenerator_t rng, float inv_temp, long long nx, long long ny,
  float j0, float j1, float j2) {

  // Setup CUDA launch configuration
  int blocks = (nx * ny + THREADS - 1) / THREADS;

  // Update black
  CHECK_CURAND(hiprandGenerateUniform(rng, randvals, nx*ny));
  update_lattice<true><<<blocks, THREADS>>>(color, lattice, randvals, inv_temp, nx, ny, j0, j1, j2);

  // Update white
  CHECK_CURAND(hiprandGenerateUniform(rng, randvals, nx*ny/2));
  update_lattice<false><<<blocks, THREADS>>>(color, lattice  , randvals, inv_temp, nx, ny, j0, j1, j2);
}

static void usage(const char *pname) {

  const char *bname = rindex(pname, '/');
  if (!bname) {bname = pname;}
  else        {bname++;}

  fprintf(stdout,
          "Usage: %s [options]\n"
          "options:\n"
          "\t-x|--lattice-n <LATTICE_N>\n"
          "\t\tnumber of lattice rows\n"
          "\n"
          "\t-y|--lattice_m <LATTICE_M>\n"
          "\t\tnumber of lattice columns\n"
          "\n"
          "\t-w|--nwarmup <NWARMUP>\n"
          "\t\tnumber of warmup iterations\n"
          "\n"
          "\t-n|--niters <NITERS>\n"
          "\t\tnumber of trial iterations\n"
          "\n"
          "\t-a|--alpha <ALPHA>\n"
          "\t\tcoefficient of critical temperature\n"
          "\n"
          "\t-s|--seed <SEED>\n"
          "\t\tseed for random number generation\n"
          "\n"
          "\t-o|--write-lattice\n"
          "\t\twrite final lattice configuration to file\n\n",
          bname);
  exit(EXIT_SUCCESS);
}

void write_info(float total_energy[], float total_energy_v, float av_energy, float variance, int niters) {
    FILE *fptr = fopen("energias.txt", "w");
    for (int i = 0; i < niters; i++) {
        fprintf(fptr, "%f \n", total_energy[i]);
    }
    fclose(fptr);
    fptr = fopen("detalhes.txt", "w");
    fprintf(fptr, "\n");
    fprintf(fptr, "Soma das energias de todas as iterações: %f\n", total_energy_v);
    fprintf(fptr, "Energia média: %f\n", av_energy);
    fprintf(fptr, "Variância: %f \n", variance);
    fclose(fptr);
}

void write_values(char* filename, float t, float sh) {
    FILE *fptr3 = fopen(filename, "a");
    fprintf(fptr3, "%f, %f ", t,  sh);
    fprintf(fptr3, "\n");
    fclose(fptr3);
}

int simulate(float alpha, float t, char* fileName, int ny, int niters) {
  // Defaults
  long long nx = 5120;
  // long long ny = 5120;
  // float alpha = 0.1f;
  int nwarmup = 100;
  // int niters = 1000;
  bool write = false;
  unsigned long long seed = 1234ULL;
  float j0, j1, j2;
  j0 = 1.0;
  j1 = (1-alpha)*j0;
  j2 = -alpha*j0;

  // while (1) {
  //   static struct option long_options[] = {
  //       {     "lattice-n", required_argument, 0, 'x'},
  //       {     "lattice-m", required_argument, 0, 'y'},
  //       {         "alpha", required_argument, 0, 'y'},
  //       {          "seed", required_argument, 0, 's'},
  //       {       "nwarmup", required_argument, 0, 'w'},
  //       {        "niters", required_argument, 0, 'n'},
  //       { "write-lattice",       no_argument, 0, 'o'},
  //       {          "help",       no_argument, 0, 'h'},
  //       {               0,                 0, 0,   0}
  //   };

  //   int option_index = 0;
  //   int ch = getopt_long(argc, argv, "x:y:a:s:w:n:oh", long_options, &option_index);
  //   if (ch == -1) break;

  //   switch(ch) {
  //     case 0:
  //       break;
  //     case 'x':
  //       nx = atoll(optarg); break;
  //     case 'y':
  //       ny = atoll(optarg); break;
  //     case 'a':
  //       alpha = atof(optarg); break;
  //     case 's':
  //       seed = atoll(optarg); break;
  //     case 'w':
  //       nwarmup = atoi(optarg); break;
  //     case 'n':
  //       niters = atoi(optarg); break;
  //     case 'o':
  //       write = true; break;
  //     case 'h':
  //       usage(argv[0]); break;
  //     case '?':
  //       exit(EXIT_FAILURE);
  //     default:
  //       fprintf(stderr, "unknown option: %c\n", ch);
  //       exit(EXIT_FAILURE);
  //   }
  // }

  // Check arguments
  if (nx % 2 != 0 || ny % 2 != 0) {
    fprintf(stderr, "ERROR: Lattice dimensions must be even values.\n");
    exit(EXIT_FAILURE);
  }

  // float inv_temp = 1.0f / (alpha*TCRIT);
  float inv_temp = 1.0f / t;

  // Setup cuRAND generator
  hiprandGenerator_t rng;
  CHECK_CURAND(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10));
  CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed(rng, seed));
  float *randvals;
  CHECK_CUDA(hipMalloc(&randvals, nx * ny * sizeof(*randvals)));

  // Setup black and white lattice arrays on device
  signed char *lattice;
  float *spin_energy;
  CHECK_CUDA(hipMalloc(&spin_energy, nx*ny * sizeof(*spin_energy)));
  CHECK_CUDA(hipMalloc(&lattice, nx * ny * sizeof(*lattice)));

  int blocks = (nx * ny + THREADS - 1) / THREADS;
  CHECK_CURAND(hiprandGenerateUniform(rng, randvals, nx*ny));
  init_spins<<<blocks, THREADS>>>(lattice, randvals, nx, ny);
  // calculate_spin_energy<<<blocks, THREADS>>>(lattice, spin_energy, nx, ny, j0, j1, j2);
  float total_energy[niters];

  // float* squareOfDistanceToMean;
  // CHECK_CUDA(hipMalloc(&squareOfDistanceToMean, niters*sizeof(float)));
  clock_t start, end;
  start = clock();
  // Warmup iterations
  printf("Starting warmup...\n");
  for (int i = 0; i < nwarmup; i++) {
    update(Color::BLACK, lattice, randvals, rng, inv_temp, nx, ny, j0, j1, j2);
    update(Color::WHITE, lattice, randvals, rng, inv_temp, nx, ny, j0, j1, j2);
    update(Color::GREEN, lattice, randvals, rng, inv_temp, nx, ny, j0, j1, j2);
  }

  CHECK_CUDA(hipDeviceSynchronize());

  printf("Starting trial iterations...\n");
  auto t0 = std::chrono::high_resolution_clock::now();
  float av_energy = 0;
  for (int i = 0; i < niters; i++) {
    update(Color::BLACK, lattice, randvals, rng, inv_temp, nx, ny, j0, j1, j2);
    update(Color::WHITE, lattice, randvals, rng, inv_temp, nx, ny, j0, j1, j2);
    update(Color::GREEN, lattice, randvals, rng, inv_temp, nx, ny, j0, j1, j2);
    calculate_spin_energy<<<blocks,THREADS>>>(lattice, spin_energy, nx, ny, j0, j1, j2);

    CHECK_CUDA(hipDeviceSynchronize());
    double* devsum;
    int nchunks = (nx * ny + CUB_CHUNK_SIZE - 1)/ CUB_CHUNK_SIZE;
    CHECK_CUDA(hipMalloc(&devsum,  nchunks * sizeof(*devsum)));
    size_t cub_workspace_bytes = 0;
    void* workspace = NULL;
    CHECK_CUDA(hipcub::DeviceReduce::Sum(workspace, cub_workspace_bytes, spin_energy, devsum, CUB_CHUNK_SIZE));
    CHECK_CUDA(hipMalloc(&workspace, cub_workspace_bytes));
    for (int j = 0; j < nchunks; j++) {
      CHECK_CUDA(hipcub::DeviceReduce::Sum(workspace, cub_workspace_bytes, &spin_energy[j*CUB_CHUNK_SIZE], devsum + j,
                              std::min((long long) CUB_CHUNK_SIZE, nx * ny - j * CUB_CHUNK_SIZE)));
    }

    double* hostsum;
    hostsum = (double*)malloc(nchunks * sizeof(*hostsum));
    CHECK_CUDA(hipMemcpy(hostsum, devsum, nchunks * sizeof(*devsum), hipMemcpyDeviceToHost));
    double fullsum = 0.0;
    for (int j = 0; j < nchunks; j++) {
      fullsum += hostsum[j];
    }
    
    CHECK_CUDA(hipFree(devsum));
    CHECK_CUDA(hipFree(workspace));
    CHECK_CUDA(hipFree(hostsum));
    total_energy[i] = fullsum;
    av_energy += fullsum;
    if (i % 1000 == 0) printf("Completed %d/%d iterations...\n", i+1, niters);
  }
  av_energy /= niters;
  float variance = 0;
  for (int i = 0; i < niters; i++) {
    variance += (total_energy[i]-av_energy)*(total_energy[i]-av_energy);
  }
  variance /= niters;
  float specific_heat = variance / (t*t*nx*ny);

  


  write_info(total_energy, av_energy * niters, av_energy, variance, niters);
  write_values(fileName, t, specific_heat);
  end = clock();
  double time_taken = ((end-start)+0.0) / CLOCKS_PER_SEC;

  FILE *fptr4 = fopen("time_taken.txt", "a");
  fprintf(fptr4, "%f, %f ", t,  specific_heat);
  fprintf(fptr4, "%f sec", time_taken);
  fprintf(fptr4, "\n");
  fclose(fptr4);
  auto t1 = std::chrono::high_resolution_clock::now();

  double duration = (double) std::chrono::duration_cast<std::chrono::microseconds>(t1-t0).count();
  printf("REPORT:\n");
  printf("\tnGPUs: %d\n", 1);
  printf("\ttemperature: %f * %f\n", alpha, TCRIT);
  printf("\tseed: %llu\n", seed);
  printf("\twarmup iterations: %d\n", nwarmup);
  printf("\ttrial iterations: %d\n", niters);
  printf("\tlattice dimensions: %lld x %lld\n", nx, ny);
  printf("\telapsed time: %f sec\n", duration * 1e-6);
  printf("\tupdates per ns: %f\n", (double) (nx * ny) * niters / duration * 1e-3);

  // Reduce
  double* devsum;
  int nchunks = (nx * ny + CUB_CHUNK_SIZE - 1)/ CUB_CHUNK_SIZE;
  CHECK_CUDA(hipMalloc(&devsum,  nchunks * sizeof(*devsum)));
  int cub_workspace_bytes = 0;
  void* workspace = NULL;
  CHECK_CUDA(hipcub::DeviceReduce::Sum(workspace, cub_workspace_bytes, lattice, devsum, CUB_CHUNK_SIZE));
  CHECK_CUDA(hipMalloc(&workspace, cub_workspace_bytes));
  for (int i = 0; i < nchunks; i++) {
    CHECK_CUDA(hipcub::DeviceReduce::Sum(workspace, cub_workspace_bytes, &lattice[i*CUB_CHUNK_SIZE], devsum + i,
                           std::min((long long) CUB_CHUNK_SIZE, nx * ny - i * CUB_CHUNK_SIZE)));
  }

  double* hostsum;
  hostsum = (double*)malloc(nchunks * sizeof(*hostsum));
  CHECK_CUDA(hipMemcpy(hostsum, devsum, nchunks * sizeof(*devsum), hipMemcpyDeviceToHost));
  double fullsum = 0.0;
  for (int i = 0; i < nchunks; i++) {
    fullsum += hostsum[i];
  }
  std::cout << "\taverage magnetism (absolute): " << abs(fullsum / (nx * ny)) << std::endl;

  // if (write) write_lattice(lattice_b, lattice_w, "final.txt", nx, ny);

  return 0;
}

int main(int argc, char* argv[]) {
  float alpha = atof(argv[1]);
  float t = atof(argv[2]);
  char* fileName = argv[3];
  int C = atoi(argv[4]);
  int iterations = atoi(argv[5]);
  simulate(alpha, t, fileName, C, iterations);
}
